#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <malloc.h>
#include <stdio.h>
#include <cmath>
#include <time.h>

//using namespace std;
//#define dimension_size 32

__global__ void add(unsigned int ** z,int a,int c, int b,int d)
{
	unsigned short pre_row = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned short pre_col = threadIdx.y + blockIdx.y * blockDim.y;
	z[pre_row][pre_col] = 0;//һ��ʼ���Ǹ�z[]����ֵ������
	unsigned short row = pre_row + c;
	unsigned short col = pre_col + b;
	//printf("%d,%d\n", pre_row, pre_col);
	if (row >=c && row <= a && col >= b && col <= d) {
		for (int i = 0; i < sizeof(row) * CHAR_BIT; i++) {
			z[pre_row][pre_col] |= (row & 1U << i) << (i + 1) | (col & 1U << i) << i;
		}
	}
}
void encode()
{
	printf("coordinate input:\n");
	unsigned int row, col,z=0;
	scanf("%d%d", &row, &col);
	for (int i = 0; i < sizeof(row) * CHAR_BIT; i++) {
		z|= (row & 1U << i) << (i + 1) | (col & 1U << i) << i;
	}
	printf("%d\n", z);
}
void decode()
{
	unsigned int z;
	int flag=0, i=0, j=0;
	int row=0, col=0;
	printf("SFC_value input:\n");
	scanf("%d", &z);
	unsigned int a[16] = { 0 };//��
	unsigned int b[16] = { 0 };//��
	while (z>0)
	{
		if (flag == 0) {
			b[i] = z % 2;
			i = i + 1;
			z = z / 2;
			flag = 1;
		}
		else {
			a[j] = z % 2;
			j = j + 1;
			z = z / 2;
			flag = 0;
		}
	}
	while (j > 0) {
		if (a[--j] == 1) {
			row += pow(2,j);
		}	
	}
	while (i > 0) {
		if (b[--i] == 1) {
			col += pow(2, i);
		}
	}
	printf("%d,%d\n", row, col);
}
void query()
{
	
	int a, b, c, d;
	printf("box input:\n");
	scanf("%d%d%d%d", &a, &b, &c, &d);
	printf("In process\n");
	clock_t start_time = clock();
	int N = a - c + 1;
	int M = d - b + 1;
	//int number = N * M;
	//int nBytes = number*sizeof(unsigned int);
	unsigned int **z;
	unsigned int *h_c;
	unsigned int **d_z;
	unsigned int *d_c;
	hipMalloc((void**)&d_z, N * sizeof(unsigned int*));
	hipMalloc((void**)&d_c, N * M * sizeof(unsigned int));
	z = (unsigned int**)malloc(N * sizeof(unsigned int*));
	h_c = (unsigned int*)malloc(N*M * sizeof(unsigned int));
	for (int r = 0; r < N; r++)
	{
		z[r] = d_c + r*M;
	}


	hipMemcpy((void*)d_z, (void*)z, N * sizeof(unsigned int*), hipMemcpyHostToDevice);
	int BLOCKCOLS = 1;
	int BLOCKROWS = 1;
	int gridCols = (M + BLOCKCOLS - 1) / BLOCKCOLS;
	int gridRows = (N + BLOCKROWS - 1) / BLOCKROWS;
	dim3 gridSize(gridCols, gridRows);
	dim3 blockSize(BLOCKCOLS, BLOCKROWS);

	//dim3 blockSize(1,16);
	//dim3 gridSize((number + blockSize.x*blockSize.y - 1) / (blockSize.x*blockSize.y));
	//add << <gridSize, blockSize >> >(d_z);
	add << <gridSize, blockSize >> >(d_z, a, c, b, d);
	//add << <1, blockSize >> >(d_z, a, c, b, d);
	hipMemcpy((void*)h_c, (void*)d_c, N*M * sizeof(unsigned int), hipMemcpyDeviceToHost);
	FILE *outfile;
	outfile = fopen("SFC_z.txt", "w");
	if (outfile == NULL) {
		printf("�޷����ļ�\n");
	}
	for (int row = 0; row < N; row++)
	{
		for (int col = 0; col < M; col++)
		{
			fprintf(outfile, "%d ", h_c[row*M + col]);
		}
		fprintf(outfile, "\n");
	}
	fclose(outfile);
	printf("finished!\n");
	clock_t end_time = clock();
	float clockTime = static_cast<double>(end_time - start_time) / CLOCKS_PER_SEC * 1000;
	printf("Running time is:   %3.2f ms\n", clockTime);
	hipFree(d_z);
	hipFree(d_c);
	free(z);
	free(h_c);
}
int main()
{
	int option;
	while (1)
	{
		printf("1.encode   2.decode  3.query\n");
		printf("please input option��");
		scanf("%d", &option);
		switch (option)
		{
		case 1:encode();
			break;
		case 2:decode();
			break;
		case 3:query();
		}
	}
	return 0;
}
