#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <malloc.h>
#include <stdio.h>
#include <cmath>
#include <time.h>
#include <thrust/sort.h>
#include <vector>

using std::vector;
__global__ void en(unsigned int * z, int N,int M)
{
	unsigned short row = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned short col = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int k = 0;//һ��ʼ���Ǹ�z[]����ֵ������һ����M������
	if (row < N && col < M) {
		for (int i = 0; i < sizeof(row) * CHAR_BIT; i++) {
			k |= (row & 1U << i) << (i + 1) | (col & 1U << i) << i;
			z[M*row + col] = k;
		}
	}
}
void encode()
{
	int N,M;
	printf("input row col:\n");
	scanf("%d%d", &N,&M);
	printf("In process\n");
	clock_t start_time = clock();
	int number = N * M;
	unsigned int *z;
	unsigned int *d_z;
	hipMalloc((void**)&d_z, number * sizeof(unsigned int));
	z = (unsigned int*)malloc(number * sizeof(unsigned int));
	hipMemcpy((void*)d_z, (void*)z, number * sizeof(unsigned int), hipMemcpyHostToDevice);
	int BLOCKCOLS = 16;
	int BLOCKROWS = 16;
	int gridCols = (M + BLOCKCOLS - 1) / BLOCKCOLS;
	int gridRows = (N + BLOCKROWS - 1) / BLOCKROWS;
	dim3 gridSize(gridRows, gridCols);//���в��ܷ��������ں˺����м������б�ǻ����
	dim3 blockSize(BLOCKROWS, BLOCKCOLS);
	en << <gridSize, blockSize >> >(d_z, N,M);
	hipMemcpy((void*)z, (void*)d_z, N*M * sizeof(unsigned int), hipMemcpyDeviceToHost);
	FILE *outfile;
	outfile = fopen("SFC_z.txt", "w");
	if (outfile == NULL) {
		printf("�޷����ļ�\n");
	}
	for (int row = 0; row < N; row++)
	{
		for (int col = 0; col < M; col++)
		{
			fprintf(outfile, "%d ", z[M*row + col]);
		}
		fprintf(outfile, "\n");
	}
	fclose(outfile);
	printf("finished!\n");
	clock_t end_time = clock();
	float clockTime = static_cast<double>(end_time - start_time) / CLOCKS_PER_SEC * 1000;
	printf("Running time is:   %3.2f ms\n", clockTime);
	hipFree(d_z);
	free(z);
}

__global__ void de(unsigned int * z, unsigned int * a, unsigned int * b,int row,int col)
{
	unsigned short pre_row = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned short pre_col = threadIdx.y + blockIdx.y * blockDim.y;
	printf("%d", pre_row);
	int flag = 0;
	int i, j;
	unsigned int m[16] = { 0 };//��
	unsigned int n[16] = { 0 };//��
	while (z[pre_row*row + pre_col]>0)
	{
		if (flag == 0) {
			n[i] = z[pre_row*row + pre_col] % 2;
			i = i + 1;
			z[pre_row*row + pre_col] = z[pre_row*row + pre_col] / 2;
			flag = 1;
		}
		else {
			m[j] = z[pre_row*row + pre_col] % 2;
			j = j + 1;
			z[pre_row*row + pre_col] = z[pre_row*row + pre_col] / 2;
			flag = 0;
		}
	}
	while (j > 0) {
		if (m[--j] == 1) {
			int x = j;
			int mul=1;
			while (x > 0) {
				mul = 2*mul;
				x--;
			}
			a[pre_row] += mul;
		}
	}
	while (i > 0) {
		if (n[--i] == 1) {
			int y = i;
			int mul2 = 1;
			while (y > 0) {
				mul2 = 2 * mul2;
				y--;
			}
			b[pre_col] += mul2;
		}
	}
}

void decode()
{
	int row = 1000;
	int col = 1000;
	int number = row * col;
	unsigned int *z;
	z = (unsigned int*)malloc(number * sizeof(unsigned int));
	unsigned int *a;
	a = (unsigned int*)malloc(row * sizeof(unsigned int));
	unsigned int *b;
	b = (unsigned int*)malloc(col * sizeof(unsigned int));
	unsigned int *d_a;
	hipMalloc((void**)&d_a, row * sizeof(unsigned int));
	unsigned int *d_b;
	hipMalloc((void**)&d_b, col * sizeof(unsigned int));
	unsigned int *d_z;
	hipMalloc((void**)&d_z, number * sizeof(unsigned int));
	
	int i, j;
	FILE *fp;
	char infile[10];
	printf("SFC_value input:\n");
	scanf("%s", infile);
	fp = fopen(infile, "r");
	if (fp == NULL)
	{
		printf("cannot open file\n");
		return;
	}
	for (i = 0; i<row; i++)
	{
		for (j = 0; j<col; j++)
		{
			fscanf(fp, "%d ", &z[i*col + j]);
		}
		fscanf(fp, "\n");
	}
	hipMemcpy((void*)d_a, (void*)a, row * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy((void*)d_b, (void*)b, col * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy((void*)d_z, (void*)z, number * sizeof(unsigned int), hipMemcpyHostToDevice);
	int BLOCKCOLS = 16;
	int BLOCKROWS = 16;
	int gridCols = (col + BLOCKCOLS - 1) / BLOCKCOLS;
	int gridRows = (row + BLOCKROWS - 1) / BLOCKROWS;
	dim3 gridSize(gridRows, gridCols);//���в��ܷ��������ں˺����м������б�ǻ����
	dim3 blockSize(BLOCKROWS, BLOCKCOLS);
	//dim3 gridSize((number + blockSize.x*blockSize.y - 1) / (blockSize.x*blockSize.y));
	//add << <gridSize, blockSize >> >(d_z);
	de << <gridSize, blockSize >> >(d_z,d_a,d_b,row,col);
	//add << <1, blockSize >> >(d_z, a, c, b, d);
	hipMemcpy((void*)a, (void*)d_a, row * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy((void*)b, (void*)d_b, row * sizeof(unsigned int), hipMemcpyDeviceToHost);
	FILE *outfile;
	outfile = fopen("decode.txt", "w");
	if (outfile == NULL) {
		printf("�޷����ļ�\n");
	}
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < col; j++)
		{
			fprintf(outfile, "(%d,%d)", a[i],b[j]);
		}
		fprintf(outfile, "\n");
	}

}

__global__ void add(unsigned int * z, int a, int c, int b, int d, int M)
{
	unsigned short pre_row = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned short pre_col = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int k = 0;//һ��ʼ���Ǹ�z[]����ֵ������һ����M������
	unsigned short row = pre_row + c;
	unsigned short col = pre_col + b;
	//printf("%d,%d\n", pre_row, pre_col);
	if (row >= c && row <= a && col >= b && col <= d) {
		for (int i = 0; i < sizeof(row) * CHAR_BIT; i++) {
			k |= (row & 1U << i) << (i + 1) | (col & 1U << i) << i;
			z[M*pre_row + pre_col] = k;
		}
	}
}

void result(unsigned int * z,int N,int M,int number) {
	FILE *outfile;
	outfile = fopen("Query.txt", "w");
	if (outfile == NULL) {
		printf("�޷����ļ�\n");
	}
	for (int i = 0; i<N; i++)
	{
		for (int j = 0; j<M; j++)
		{
			fprintf(outfile, "%d ", z[i*M + j]);
		}
		fprintf(outfile, "\n");
	}
	fprintf(outfile, "\n");
	thrust::sort(z, z + number);//����
	int max = number - 1;
	vector<unsigned int> rangevec;
	int nstart = 0;
	int i = 0; //current pos
	while (1)
	{
		i++;
		if (i == max)//over the last one
		{
			rangevec.push_back(z[nstart]);
			rangevec.push_back(z[i-1]);
			rangevec.push_back(z[i]);
			break;
		}
		if (z[i] != (z[i - 1] + 1)) //discontinuous
		{
			rangevec.push_back(z[nstart]);
			rangevec.push_back(z[i - 1]);
			nstart = i;
		}
	}
	for (int i = 0; i < rangevec.size(); i++)
	{
		if (i % 2 == 0) {
			fprintf(outfile, "%d", rangevec[i]);
		}
		else {
			fprintf(outfile, "-%d", rangevec[i]);
			fprintf(outfile, " ");
		}
	}
	fclose(outfile);
}
void query()
{
	int a, b, c, d;
	printf("box input:\n");
	scanf("%d%d%d%d", &a, &b, &c, &d);
	printf("In process\n");
	clock_t start_time = clock();
	int N = a - c + 1;
	int M = d - b + 1;
	
	int number = N * M;
	//int nBytes = number*sizeof(unsigned int);
	unsigned int *z;
	unsigned int *d_z;
	hipMalloc((void**)&d_z, number * sizeof(unsigned int));
	z = (unsigned int*)malloc(number * sizeof(unsigned int));
	hipMemcpy((void*)d_z, (void*)z, number * sizeof(unsigned int), hipMemcpyHostToDevice);
	int BLOCKCOLS = 16;
	int BLOCKROWS = 16;
	int gridCols = (M + BLOCKCOLS - 1) / BLOCKCOLS;
	int gridRows = (N + BLOCKROWS - 1) / BLOCKROWS;
	dim3 gridSize(gridRows,gridCols);//���в��ܷ��������ں˺����м������б�ǻ����
	dim3 blockSize(BLOCKROWS, BLOCKCOLS);
	add << <gridSize, blockSize >> >(d_z, a, c, b, d,M);
	hipMemcpy((void*)z, (void*)d_z, N*M * sizeof(unsigned int), hipMemcpyDeviceToHost);
	result(z, N, M, number);
	printf("finished!\n");
	hipFree(d_z);
	free(z);
}

int main()
{
	//decode();
	//encode();
	query();
	int option;
	while (1)
	{
		printf("1.encode   2.decode  3.query\n");
		printf("please input option��");
		scanf("%d", &option);
		switch (option)
		{
		case 1:encode();
			break;
		case 2:decode();
			break;
		case 3:query();
		}
	}
	//clock_t end_time = clock();
	//float clockTime = static_cast<double>(end_time - start_time) / CLOCKS_PER_SEC * 1000;
	//printf("Running time is:   %3.2f ms\n", clockTime);
	return 0;
}
